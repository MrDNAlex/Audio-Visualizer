#include "hip/hip_runtime.h"
#include "Convolution.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "MemoryManagement.h"
#include "lodepng.h"

int GetConvolutionOutputSize(int width, int kernelSize, int stepSize)
{
	return ((width - kernelSize) / stepSize) + 1;
}

__global__ void ConvolutionGPU(float* input, float* kernel, float* output, int* arraySize, int* kernelSize, int* step, int* outputSize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int N = *arraySize;
	int K = *kernelSize;
	int S = *step;

	if (index >= *outputSize) return;

	float sum = 0.0f;

	int startIndex = index * S;

	for (int i = 0; i < K; i++) {
		int inputIndex = startIndex + i;
		int kernelIndex = K - 1 - i;

		// Check bounds for the input array
		if (inputIndex >= N) break;

		sum += input[inputIndex] * kernel[kernelIndex];
	}

	output[index] = sum;
}

hipError_t Convolution(float* input, float* kernel, float* output, int arraySize, int kernelSize, int stepSize, int outputSize)
{
	float* gpuInput = 0;
	float* gpuKernel = 0;
	float* gpuOutput = 0;
	int* gpuArraySize = 0;
	int* gpuKernelSize = 0;
	int* gpuStep = 0;
	int* gpuOutputSize = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	cudaStatus = AssignVariable((void**)&gpuArraySize, &arraySize, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuKernelSize, &kernelSize, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuStep, &stepSize, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuOutputSize, &outputSize, sizeof(int));

	cudaStatus = AssignVariable((void**)&gpuInput, input, sizeof(float), arraySize);
	cudaStatus = AssignVariable((void**)&gpuKernel, kernel, sizeof(float), kernelSize);

	cudaStatus = AssignMemory((void**)&gpuOutput, sizeof(float), outputSize);

	int threads = 1024;
	int blocks = (outputSize / threads) + 1;

	ConvolutionGPU << <blocks, threads >> > (gpuInput, gpuKernel, gpuOutput, gpuArraySize, gpuKernelSize, gpuStep, gpuOutputSize);

	cudaStatus = hipDeviceSynchronize();

	cudaStatus = GetVariable(output, gpuOutput, sizeof(float), outputSize);

	hipFree(gpuInput);
	hipFree(gpuKernel);
	hipFree(gpuOutput);
	hipFree(gpuArraySize);
	hipFree(gpuKernelSize);
	hipFree(gpuStep);
	hipFree(gpuOutputSize);

	return cudaStatus;
}

std::pair<int, int> GetConvolutionOutputSize2D(int width, int height, int kernelWidth, int kernelHeight, int stepWidth, int stepHeight) {
	int outputWidth = GetConvolutionOutputSize(width, kernelWidth, stepWidth);
	int outputHeight = GetConvolutionOutputSize(height, kernelHeight, stepHeight);

	return { outputWidth, outputHeight };
}

__global__ void Convolution2DGPU(float* input, float* kernel, float* output, int* inputWidth, int* inputHeight, int* kernelWidth, int* kernelHeight, int* stepWidth, int* stepHeight, int* outputWidth, int* outputHeight)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	int KWidth = *kernelWidth;
	int KHeight = *kernelHeight;
	int SWidth = *stepWidth;
	int SHeight = *stepHeight;

	if (col >= *outputWidth || row >= *outputHeight) return;

	float sum = 0.0f;

	int startRow = row * SHeight;
	int startCol = col * SWidth;

	for (int i = 0; i < KHeight; ++i) {
		for (int j = 0; j < KWidth; ++j) {
			int rowIndex = startRow + i;
			int colIndex = startCol + j;

			if (rowIndex >= *inputHeight || colIndex >= *inputWidth) continue;

			int inputIndex = rowIndex * *inputWidth + colIndex;
			int kernelIndex = (KHeight - 1 - i) * KWidth + (KWidth - 1 - j);

			sum += input[inputIndex] * kernel[kernelIndex];
		}
	}

	int outputIndex = row * *outputWidth + col;
	output[outputIndex] = sum;
}

hipError_t Convolution2D(float* input, float* kernel, float* output, int inputWidth, int inputHeight, int kernelWidth, int kernelHeight, int stepWidth, int stepHeight, int outputWidth, int outputHeight)
{
	float* gpuInput = 0;
	float* gpuKernel = 0;
	float* gpuOutput = 0;
	int* gpuInputWidth = 0;
	int* gpuInputHeight = 0;
	int* gpuKernelWidth = 0;
	int* gpuKernelHeight = 0;
	int* gpuStepWidth = 0;
	int* gpuStepHeight = 0;
	int* gpuOutputWidth = 0;
	int* gpuOutputHeight = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	cudaStatus = AssignVariable((void**)&gpuInputWidth, &inputWidth, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuInputHeight, &inputHeight, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuKernelWidth, &kernelWidth, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuKernelHeight, &kernelHeight, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuStepWidth, &stepWidth, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuStepHeight, &stepHeight, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuOutputWidth, &outputWidth, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuOutputHeight, &outputHeight, sizeof(int));

	cudaStatus = AssignVariable((void**)&gpuInput, input, sizeof(float), inputWidth * inputHeight);
	cudaStatus = AssignVariable((void**)&gpuKernel, kernel, sizeof(float), kernelWidth * kernelHeight);

	cudaStatus = AssignMemory((void**)&gpuOutput, sizeof(float), outputWidth * outputHeight);

	dim3 threads(16, 16);
	dim3 blocks((outputWidth / threads.x) + 1, (outputHeight / threads.y) + 1);

	Convolution2DGPU << <blocks, threads >> > (gpuInput, gpuKernel, gpuOutput, gpuInputWidth, gpuInputHeight, gpuKernelWidth, gpuKernelHeight, gpuStepWidth, gpuStepHeight, gpuOutputWidth, gpuOutputHeight);

	cudaStatus = hipDeviceSynchronize();

	cudaStatus = GetVariable(output, gpuOutput, sizeof(float), outputWidth * outputHeight);

	hipFree(gpuInput);
	hipFree(gpuKernel);
	hipFree(gpuOutput);
	hipFree(gpuInputWidth);
	hipFree(gpuInputHeight);
	hipFree(gpuKernelWidth);
	hipFree(gpuKernelHeight);
	hipFree(gpuStepWidth);
	hipFree(gpuStepHeight);
	hipFree(gpuOutputWidth);
	hipFree(gpuOutputHeight);

	return cudaStatus;
}

__global__ void drawKernel(unsigned char* img, int width, int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < width && y < height) {
		int offset = (y * width + x) * 4;
		img[offset] = x % 256;  // Red channel example
		img[offset + 1] = y % 256; // Green channel example
		img[offset + 2] = 128; // Blue channel static example
		img[offset + 3] = 20; // Alpha channel
	}
}

void DrawImage(int index) {
	int width = 1024;
	int height = 1024;
	size_t img_size = width * height * 4;
	unsigned char* d_img, * h_img;

	hipMalloc(&d_img, img_size);
	h_img = (unsigned char*)malloc(img_size);

	dim3 blockSize(16, 16);
	dim3 numBlocks((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);
	drawKernel << <numBlocks, blockSize >> > (d_img, width, height);
	hipMemcpy(h_img, d_img, img_size, hipMemcpyDeviceToHost);

	char filename[100];

	// Format the filename with the index
	sprintf(filename, "C:\\Users\\MrDNA\\Downloads\\test\\output%d.png", index);
	
	// Save to PNG using lodepng
	unsigned error = lodepng_encode32_file(filename, h_img, width, height);
	if (error) printf("Error %u: %s\n", error, lodepng_error_text(error));

	hipFree(d_img);
	free(h_img);
}
