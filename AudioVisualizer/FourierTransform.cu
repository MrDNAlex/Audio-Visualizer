#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "MemoryManagement.h"
#include "FourierTransform.cuh"
#include <iostream>
#include <chrono>

__global__ void DFTMagnitudeGPU(float* input, float* output, int* fft_size, int* numOfFrames)
{
	const float pi = 3.14159265358979323846;
	int totalThreads = blockDim.x * gridDim.x;
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	int fftSize = *fft_size;
	int frameNums = *numOfFrames;

	int totalOps = fftSize * frameNums;

	if (threadId < totalOps) {
		int frameIndex = threadId / fftSize;  
		int k = threadId % fftSize;

		float real = 0.0f;
		float imag = 0.0f;
		float angleStart = 2 * pi * k / fftSize;

		for (int n = 0; n < fftSize; n++)
		{
			float angle = angleStart * n;
			float inputValue = input[frameIndex * fftSize + n];

			real += inputValue * cosf(angle);
			imag += inputValue * sinf(angle);
		}

		output[frameIndex * fftSize + k] = sqrtf(real * real + imag * imag);
	}
}

hipError_t FourierTransformMagnitude(float* input, float* output, int fft_size, int numOfFrames)
{
	std::cout << "FourierTransform" << std::endl;

	int signalSize = fft_size * numOfFrames;

	float* kernel_input = 0;
	float* kernel_output = 0;
	int* kernel_fft_size = 0;
	int* kernel_numOfFrames = 0;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	cudaStatus = AssignVariable((void**)&kernel_input, input, sizeof(float), signalSize);
	cudaStatus = AssignVariable((void**)&kernel_fft_size, &fft_size, sizeof(int));
	cudaStatus = AssignVariable((void**)&kernel_numOfFrames, &numOfFrames, sizeof(int));

	cudaStatus = AssignMemory((void**)&kernel_output, sizeof(float), signalSize);

	int threadsPerBlock = 1024;
	int totalOps = fft_size * numOfFrames;
	int blocks = (totalOps + threadsPerBlock - 1) / threadsPerBlock;

	DFTMagnitudeGPU << <blocks, threadsPerBlock >> > (kernel_input, kernel_output, kernel_fft_size, kernel_numOfFrames);

	cudaStatus = hipDeviceSynchronize();

	cudaStatus = GetVariable(output, kernel_output, sizeof(float), signalSize);

	hipFree(kernel_input);
	hipFree(kernel_output);
	hipFree(kernel_fft_size);
	hipFree(kernel_numOfFrames);

	std::cout << "Finished FourierTransform" << std::endl;

	return cudaStatus;
}	

__global__ void DFTGPU(float* input, FourierData* output, int* fft_size, int* numOfFrames)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int N = *fft_size;
	int outputIndex = N * index;
	const float pi = 3.14159265358979323846;

	if (index >= *numOfFrames * *fft_size) return;

	for (int k = 0; k < N; k++)
	{
		float real = 0.0f;
		float imag = 0.0f;
		float angleStart = 2 * pi * k / N;

		for (int n = 0; n < N; n++)
		{
			float angle = angleStart * n;
			float inputValue = input[outputIndex + n];

			real += inputValue * cosf(angle);
			imag += inputValue * sinf(angle);
		}

		int signalIndex = outputIndex + k;

		output[signalIndex].real = real;
		output[signalIndex].imag = imag;
	}
}

hipError_t FourierTransform(float* input, FourierData* output, int fft_size, int numOfFrames)
{
	std::cout << "FourierTransform" << std::endl;

	int signalSize = fft_size * numOfFrames;

	float* kernel_input = 0;
	FourierData* kernel_output = 0;
	int* kernel_fft_size = 0;
	int* kernel_numOfFrames = 0;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	cudaStatus = AssignVariable((void**)&kernel_input, input, sizeof(float), signalSize);
	cudaStatus = AssignVariable((void**)&kernel_fft_size, &fft_size, sizeof(int));
	cudaStatus = AssignVariable((void**)&kernel_numOfFrames, &numOfFrames, sizeof(int));

	cudaStatus = AssignMemory((void**)&kernel_output, sizeof(FourierData), signalSize);

	int threads = 1024;
	int blocks = (numOfFrames / threads) + 1;

	DFTGPU << <blocks, threads >> > (kernel_input, kernel_output, kernel_fft_size, kernel_numOfFrames);

	cudaStatus = hipDeviceSynchronize();

	cudaStatus = GetVariable(output, kernel_output, sizeof(float), signalSize);

	hipFree(kernel_input);
	hipFree(kernel_output);
	hipFree(kernel_fft_size);
	hipFree(kernel_numOfFrames);

	std::cout << "Finished FourierTransform" << std::endl;

	return cudaStatus;
}