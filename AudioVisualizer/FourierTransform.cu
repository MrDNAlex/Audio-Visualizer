#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "MemoryManagement.h"
#include "FourierTransform.cuh"
#include <iostream>
#include <chrono>

__global__ void DFTMagnitudeGPU(float* input, float* output, int* fft_size, int* numOfFrames)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int N = *fft_size;
	int outputIndex = N * index;
	const float pi = 3.14159265358979323846;

	if (index >= *numOfFrames) return;

	for (int k = 0; k < N; k++)
	{
		float real = 0.0f;
		float imag = 0.0f;
		float angleStart = 2 * pi * k / N;

		for (int n = 0; n < N; n++)
		{
			float angle = angleStart * n;
			float inputValue = input[outputIndex + n];

			real += inputValue * cosf(angle);
			imag += inputValue * sinf(angle);
		}

		int signalIndex = outputIndex + k;

		output[signalIndex] = sqrtf(real * real + imag * imag);
	}
}

__global__ void DFTGPU(float* input, FourierData* output, int* fft_size, int* numOfFrames)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int N = *fft_size;
	int outputIndex = N * index;
	const float pi = 3.14159265358979323846;

	if (index >= *numOfFrames) return;

	for (int k = 0; k < N; k++)
	{
		float real = 0.0f;
		float imag = 0.0f;
		float angleStart = 2 * pi * k / N;

		for (int n = 0; n < N; n++)
		{
			float angle = angleStart * n;
			float inputValue = input[outputIndex + n];

			real += inputValue * cosf(angle);
			imag += inputValue * sinf(angle);
		}

		int signalIndex = outputIndex + k;

		output[signalIndex].real = real;
		output[signalIndex].imag = imag;
	}
}

hipError_t FourierTransformMagnitude(float* input, float* output, int fft_size, int numOfFrames)
{
	std::cout << "FourierTransform" << std::endl;

	int signalSize = fft_size * numOfFrames;

	float* kernel_input = 0;
	float* kernel_output = 0;
	int* kernel_fft_size = 0;
	int* kernel_numOfFrames = 0;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	cudaStatus = AssignVariable((void**)&kernel_input, input, sizeof(float), signalSize);
	cudaStatus = AssignVariable((void**)&kernel_fft_size, &fft_size, sizeof(int));
	cudaStatus = AssignVariable((void**)&kernel_numOfFrames, &numOfFrames, sizeof(int));

	cudaStatus = AssignMemory((void**)&kernel_output, sizeof(float), signalSize);

	int threads = 1024;
	int blocks = (numOfFrames / threads) + 1;

	DFTMagnitudeGPU << <blocks, threads >> > (kernel_input, kernel_output, kernel_fft_size, kernel_numOfFrames);

	cudaStatus = hipDeviceSynchronize();

	cudaStatus = GetVariable(output, kernel_output, signalSize);

	hipFree(kernel_input);
	hipFree(kernel_output);
	hipFree(kernel_fft_size);
	hipFree(kernel_numOfFrames);

	return cudaStatus;
}


hipError_t FourierTransform(float* input, FourierData* output, int fft_size, int numOfFrames)
{
	std::cout << "FourierTransform" << std::endl;

	int signalSize = fft_size * numOfFrames;

	float* kernel_input = 0;
	FourierData* kernel_output = 0;
	int* kernel_fft_size = 0;
	int* kernel_numOfFrames = 0;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	cudaStatus = AssignVariable((void**)&kernel_input, input, sizeof(float), signalSize);
	cudaStatus = AssignVariable((void**)&kernel_fft_size, &fft_size, sizeof(int));
	cudaStatus = AssignVariable((void**)&kernel_numOfFrames, &numOfFrames, sizeof(int));

	cudaStatus = AssignMemory((void**)&kernel_output, sizeof(FourierData), signalSize);

	int threads = 1024;
	int blocks = (numOfFrames / threads) + 1;

	DFTGPU << <blocks, threads >> > (kernel_input, kernel_output, kernel_fft_size, kernel_numOfFrames);

	cudaStatus = hipDeviceSynchronize();

	cudaStatus = GetVariable(output, kernel_output, signalSize);

	hipFree(kernel_input);
	hipFree(kernel_output);
	hipFree(kernel_fft_size);
	hipFree(kernel_numOfFrames);

	return cudaStatus;
}