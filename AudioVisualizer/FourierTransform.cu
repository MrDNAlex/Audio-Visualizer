#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "MemoryManagement.cpp"
#include "FourierTransform.cuh"
#include <iostream>

__global__ void DFTGPU(float* input, float* output_real, float* output_imag, int* fft_size, int* numOfFrames)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int N = *fft_size;
	int outputIndex = N * index;
	const float pi = 3.14159265358979323846;

	if (index >= *numOfFrames) return;

	for (int k = 0; k < N; k++)
	{
		float real = 0.0f;
		float imag = 0.0f;

		for (int n = 0; n < N; n++) {

			float angle = 2 * pi * k * n / N;

			float inputValue = input[outputIndex + n];

			real += inputValue * cosf(angle);
			imag += inputValue * sinf(angle);
		}

		output_real[outputIndex + k] = real;
		output_imag[outputIndex + k] = imag;
	}
}

hipError_t FourierTransform(float* input, float* output_real, float* output_imag, int fft_size, int numOfFrames)
{
	std::cout << "FourierTransform" << std::endl;

	int signalSize = fft_size * numOfFrames;

	float* kernel_input = 0;
	float* kernel_output_imag = 0;
	float* kernel_output_real = 0;
	int* kernel_fft_size = 0;
	int* kernel_numOfFrames = 0;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	cudaStatus = AssignVariable((void**)&kernel_input, input, sizeof(float), signalSize);
	cudaStatus = AssignVariable((void**)&kernel_fft_size, &fft_size, sizeof(int));
	cudaStatus = AssignVariable((void**)&kernel_numOfFrames, &numOfFrames, sizeof(int));

	cudaStatus = AssignMemory((void**)&kernel_output_real, sizeof(float), signalSize);
	cudaStatus = AssignMemory((void**)&kernel_output_imag, sizeof(float), signalSize);

	int threads = 1024;
	int blocks = (numOfFrames / 1024) + 1;

	DFTGPU << <blocks, threads >> > (kernel_input, kernel_output_real, kernel_output_imag, kernel_fft_size, kernel_numOfFrames);

	cudaStatus = hipDeviceSynchronize();

	//output_real = new float[signalSize];
	//output_imag = new float[signalSize];

	cudaStatus = GetVariable(output_real, kernel_output_real, signalSize);
	cudaStatus = GetVariable(output_imag, kernel_output_imag, signalSize);

	hipFree(kernel_input);
	hipFree(kernel_output_real);
	hipFree(kernel_output_imag);
	hipFree(kernel_fft_size);
	hipFree(kernel_numOfFrames);



	return cudaStatus;

}